#include "hip/hip_runtime.h"
#include <stdlib.h>    

struct RandomWalkApp {
  __host__ __device__ int steps() {return 10;}

  __host__ __device__ 
  int stepSize(int k) {
    return 1;
  }

  __host__ __device__ int samplingType()
  {
    return SamplingType::IndividualNeighborhood;
  }

  __host__ __device__ OutputFormat outputFormat()
  {
    return SampledVertices;
  }

  #define VERTICES_PER_SAMPLE 1

  __host__ __device__ EdgePos_t numSamples(CSR* graph)
  {
    //return graph->get_n_vertices() < 256*1024 ? 100 * graph->get_n_vertices() : graph->get_n_vertices();
    //return 100000000;
    return graph->get_n_vertices() * 2;
  }

  template<class SampleType>
  __host__ std::vector<VertexID_t> initialSample(int sampleIdx, CSR* graph, SampleType& sample)
  {
    std::vector<VertexID_t> initialValue;

    for (int i = 0; i < VERTICES_PER_SAMPLE; i++) {
      initialValue.push_back(sampleIdx%graph->get_n_vertices());
    }

    return initialValue;
  }

  __host__ __device__ EdgePos_t initialSampleSize(CSR* graph)
  {
    return VERTICES_PER_SAMPLE;
  }

  __host__ __device__ bool hasExplicitTransits()
  {
    return false;
  }

  template<class SampleType>
  __host__ __device__ VertexID_t stepTransits(int step, const VertexID_t sampleID, SampleType& sample, int transitIdx, hiprandState* randState)
  {
    return -1;
  }

  template<class SampleType>
  __host__ SampleType initializeSample(CSR* graph, const VertexID_t sampleID)
  {
    SampleType sample = SampleType ();

    return sample;
  }
};

struct DeepWalkApp : public RandomWalkApp {
  template<typename SampleType, typename EdgeArray, typename WeightArray>
  __device__ inline
  VertexID next(int step, CSRPartition* csr, const VertexID* transit, const VertexID sampleIdx,
                SampleType* sample, 
                const float max_weight,
                EdgeArray& transitEdges, WeightArray& transitEdgeWeights,
                const EdgePos_t numEdges, const VertexID_t neighbrID, hiprandState* state)
  {
    if (numEdges == 0) {
      return -1;
    }
    if (numEdges == 1) {
      return transitEdges[0];
    }
    
    EdgePos_t x = RandNumGen::rand_int(state, numEdges);

    /*
    float y = hiprand_uniform(state)*max_weight;

    while (y > transitEdgeWeights[x]) {
      x = RandNumGen::rand_int(state, numEdges);
      y = hiprand_uniform(state)*max_weight;
    }
    */

    return transitEdges[x];
  }
};

struct PPRApp : public RandomWalkApp {
  template<typename SampleType, typename EdgeArray, typename WeightArray>
  __device__ inline
  VertexID next(int step, CSRPartition* csr, const VertexID* transit, const VertexID sampleIdx,
                SampleType* sample, 
                const float max_weight,
                EdgeArray& transitEdges, WeightArray& transitEdgeWeights,
                const EdgePos_t numEdges, const VertexID_t neighbrID, hiprandState* state)
  {
    if (numEdges == 0) {
      return -1;
    }
    const float walkEndProb = 0.15;
    float p = hiprand_uniform(state);
    if (p < walkEndProb) {
      return -1;
    }

    if (numEdges == 1) {
      return transitEdges[0];
    }
    
    EdgePos_t x = RandNumGen::rand_int(state, numEdges);

    /*
    float y = hiprand_uniform(state)*max_weight;

    while (y > transitEdgeWeights[x]) {
      x = RandNumGen::rand_int(state, numEdges);
      y = hiprand_uniform(state)*max_weight;
    }
    */

    return transitEdges[x];
  }
};

class DummySample
{

};

struct Node2VecApp : public RandomWalkApp {
  template<typename SampleType, typename EdgeArray, typename WeightArray>
  __device__ inline
  VertexID next(int step, CSRPartition* csr, const VertexID* transits, const VertexID sampleIdx,
                SampleType* sample, 
                const float max_weight,
                EdgeArray& transitEdges, WeightArray& transitEdgeWeights,
                const EdgePos_t numEdges, const VertexID_t neighbrID, hiprandState* state)
  {
    if (numEdges == 0) {
      return -1;
    }
    if (numEdges == 1 || step == 0) {
      sample->t = *transits;
      return transitEdges[0];
    }  
    
    const float p = 2.0f;
    const float q = 0.5f;

    do {
      EdgePos_t x = RandNumGen::rand_int(state, numEdges);
      VertexID v = transitEdges[x];
      float y = hiprand_uniform(state)*max(max(p, 1/q), 1.0f);
      const CSR::Edge* tEdges = csr->get_edges(sample->t);
      EdgePos_t tNumEdges = csr->get_n_edges_for_vertex(sample->t);
      float h;
      if (x == sample->t) {
        h = p;
      } else if (utils::binarySearch(tEdges, v, tNumEdges)) {
        h = 1/q;
      } else {
        h = 1.0f;
      }

      if (y < h) {
        sample->t = *transits;
        return v;
      }
    } while (true);
  }
};

class Node2VecSample {
public:
  VertexID t;
};